#include "hip/hip_runtime.h"
// -*- C++ -*-

#include <memory>

#include "expand.H"
#include "Component.H"
#include "cudaParticle.cuH"

#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/generate.h>
#include <thrust/sequence.h>
#include <thrust/reduce.h>

unsigned Component::cudaStreamData::totalInstances=0;

using PII=std::pair<int, int>;

struct testCountLevel :  public thrust::unary_function<cudaParticle, int>
{
  int _l;

  __host__ __device__
  testCountLevel(int l) : _l(l) {}

  __host__ __device__
  int operator()(const cudaParticle& p) const
  {
    if (p.lev[0] == _l) return 1;
    return 0;
  }
};

struct testCountLevel2 :  public thrust::unary_function<int, int>
{
  int _l;

__host__ __device__
  testCountLevel2(int l) : _l(l) {}

  __host__ __device__
  int operator()(const int p) const
  {
    if (p == _l) return 1;
    return 0;
  }
};

Component::cudaStreamData::cudaStreamData()
{
  // Not sure why this breaks thrust, but it does . . .
  /*
  cuda_safe_call(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  */

  // Need blocking until thrust bug in binary search is fixed
  cuda_safe_call(hipStreamCreate(&stream),
		 __FILE__, __LINE__,
		 "Component::cudaStreamData: error creating stream");
  instance = totalInstances++;
}

Component::cudaStreamData::~cudaStreamData()
{
  cuda_safe_call(hipStreamDestroy(stream), __FILE__, __LINE__,
		 "Component::cudaStreamData: error destroying stream");
  totalInstances--;
}

void Component::cuda_initialize()
{
  cuStream = std::make_shared<cudaStreamData>();
}


Component::I2vec Component::CudaSortLevelChanges()
{
  // The plan: for the current active level search above and below for
  // particles for correction to coefficient matrix
  //
  // 1. Sort all particles by current level
  // 2. Get indices to range for each level L
  // 3. Within each level L, compute the ranges for changes,
  //    delta L = [-L, multistep-L]
  // 4. For each (L, delta L), compute the coefficient changes and
  //    apply to the appropriate coefficient matrices

  I2vec ret(multistep+1);
  for (auto & v : ret) v.resize(multistep+1);

  // Particle number
  //
  auto N = cuStream->cuda_particles.size();

  // Resize pair list and pair index
  //
  cuStream->levPair.resize(N);
  cuStream->indx2.  resize(N);

  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    thrust::device_vector<cudaParticle>::iterator
      pbeg = cuStream->cuda_particles.begin(),
      pend = cuStream->cuda_particles.end();
    
    // This gets a vector of pairs [current index, desired index],
    // leaving the order of particle structures unchanged
    //
    if (thrust_binary_search_workaround) {
      hipStreamSynchronize(cuStream->stream);
      thrust::transform(pbeg, pend, cuStream->levPair.begin(), cuPartToChange());
    } else {
      thrust::transform(exec, pbeg, pend, cuStream->levPair.begin(), cuPartToChange());
    }
    
    // Make the initial sequential index
    //
    // thrust::sequence(cuStream->indx2.begin(), cuStream->indx2.end(), 0, 1);
    thrust::sequence(cuStream->indx2.begin(), cuStream->indx2.end(), 0);
  
    // Sort the keys and make the index
    //
    thrust::sort_by_key(cuStream->levPair.begin(), cuStream->levPair.end(),
			cuStream->indx2.begin());

    // This will be [from, to] pair for determining the change matrix
    //
    thrust::pair<int, int> tr2;

    for (int target=0; target<=multistep; target++) {

      // From level is 'target'
      //
      tr2.first = target;

      for (int del=0; del<=multistep; del++) {

	// Do nothing: equal beg and end results in no level
	// processing
	//
	if (del==target) {
	  ret[target][del] = {0, 0};
	  continue;
	}
	
	// To level is 'del'
	//
	tr2.second = del;

	thrust::device_vector<thrust::pair<int, int>>::iterator
	  lbeg = cuStream->levPair.begin(), lo;

	thrust::device_vector<thrust::pair<int, int>>::iterator
	  lend = cuStream->levPair.end(),   hi;

	// Determine upper and lower indices into indx2 for the [from,
	// to] pair
	//
	if (thrust_binary_search_workaround) {
	  hipStreamSynchronize(cuStream->stream);
	  lo  = thrust::lower_bound(lbeg, lend, tr2);
	} else {
	  lo = thrust::lower_bound(exec, lbeg, lend, tr2);
	}
	
	hipStreamSynchronize(cuStream->stream);

	if (thrust_binary_search_workaround) {
	  hi = thrust::upper_bound(lbeg, lend, tr2);
	} else {
	  hi = thrust::upper_bound(exec, lbeg, lend, tr2);
	}

	cuda_safe_call_mpi(hipStreamSynchronize(cuStream->stream),
			   __FILE__, __LINE__, myid,
			   "Component::SortLevelChanges");

	ret[target][del] = {thrust::distance(lbeg, lo), 
			    thrust::distance(lbeg, hi)};
      }
    }

  }
  catch(std::bad_alloc &e) {
    std::cerr << "Ran out of memory while sorting" << std::endl;
    exit(-1);
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  // Debugging output for level changes
  //
  if (false) {
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
    std::cout << "---- " << name << " [" << myid << "] mstep=" << mstep
	      << " mdrft=" << mdrft << std::endl;
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
    for (int m1=0; m1<=multistep; m1++) {
      for (int m2=0; m2<=multistep; m2++) {
	std::cout << std::setw(15) << ret[m1][m2].second - ret[m1][m2].first;
      }
      std::cout << std::endl;
    }
    std::cout << std::string(15*(multistep+1), '-') << std::endl;
  }

  // Direct comparison with particle vector
  // [This is expensive and only for deep debugging]
  //
  if (false) {

    bool total = true;		// Print totals in matrix rather than
				// particle end points

    for (int rank=0; rank<numprocs; rank++) {
      if (rank==myid) {
	std::cout << std::string(40, '-') << std::endl
		  << "---- Sorting debug for " << name
		  << " [" << myid << "] mstep=" << mstep
		  << " T=" << tnow << std::endl
		  << std::string(40, '-') << std::endl;
	
	// Get particles from device
	//
	thrust::host_vector<cudaParticle> pp = cuStream->cuda_particles;
	thrust::host_vector<thrust::pair<int, int>> lp = cuStream->levPair;
	thrust::host_vector<int> li = cuStream->indx2;
	
	// Check particles from level lists
	//
	int good = 0, bad = 0;
	for (int i=0; i<li.size(); i++) {
	  int j = li[i];
	  if (pp[j].lev[0] != lp[i].first and pp[j].lev[1] != lp[i].second)
	    bad++;
	  else
	    good++;
	}
	std::cout << "---- Index check: good=" << good << " bad=" << bad << std::endl;

	// Make a map for checking indices
	//
	std::vector<int> lookup(li.size());
	for (int i=0; i<li.size(); i++) lookup[li[i]] = i;
	
	good = bad = 0;
	for (int i=0; i<li.size(); i++) {
	  int j = lookup[i];
	  if (pp[i].lev[0] != lp[j].first and pp[i].lev[1] != lp[j].second)
	    bad++;
	  else
	    good++;
	}
	
	std::cout << "---- Parts check: good=" << good << " bad=" << bad << std::endl;
	std::cout << std::string(40, '-') << std::endl;
	
	// Check the return matrix by simple serial pass
	//
	I2vec test(multistep+1);
	for (auto & v : test) v.resize(multistep+1, {0, 0});
	
	int l1 = 0, l2 = 0;
	test[l1][l2].first = 0;

	for (int i=0; i<lp.size(); i++) {
	  // Update list ?
	  //
	  if (lp[i].first>l1 or lp[i].second>l2) {
	    
	    if (lp[i].first>l1) {
	      // Current entry
	      test[l1][l2].second = i;
	      // Complete current 'to' level
	      for (int j2=l2+1; j2<=multistep; j2++) {
		test[l1][j2] = {i, i};
	      }

	      // Next entire from level
	      for (int j1=l1+1; j1<lp[i].first; j1++) {
		for (int j2=0; j2<=multistep; j2++) {
		  test[j1][j2] = {i, i};
		}
	      }

	      // Final to level
	      for (int j2=0; j2<lp[i].second; j2++) {
		test[lp[i].first][j2] = {i, i};
	      }
	      
	    } else if (lp[i].second>l2) {
	      for (int j2=l2+1; j2<lp[i].second; j2++) {
		test[l1][j2] = {i, i};
	      }
	    }

	    test[l1][l2].second = i;
	    test[lp[i].first][lp[i].second].first = i;
	    l1 = lp[i].first;
	    l2 = lp[i].second;
	  }
	}
	test[l1][l2].second = lp.size();
	
	std::cout << "---- Original" << std::endl;
	for (int m0=0; m0<=multistep; m0++) {
	  for (int m1=0; m1<=multistep; m1++) {
	    if (total)
	      if (m0==m1) std::cout << std::setw(10) << 0;
	      else        std::cout << std::setw(10)
				    << ret[m0][m1].second - ret[m0][m1].first;
	    else {
	      std::ostringstream sout;
	      sout << "(" << ret[m0][m1].first
		   << "," << ret[m0][m1].second << ")";
	      
	      std::cout << std::setw(20) << sout.str();
	    }
	  }
	  std::cout << std::endl;
	}
	
	std::cout << "---- Recomputed" << std::endl;
	for (int m0=0; m0<=multistep; m0++) {
	  for (int m1=0; m1<=multistep; m1++) {
	    if (total) {
	      if (m0==m1) std::cout << std::setw(10) << 0;
	      else        std::cout << std::setw(10)
				    << test[m0][m1].second - test[m0][m1].first;
	    } else {
	      std::ostringstream sout;
	      sout << "(" << test[m0][m1].first
		   << "," << test[m0][m1].second << ")";
	      
	      std::cout << std::setw(20) << sout.str();
	    }
	  }
	  std::cout << std::endl;
	}

	int differ = 0, checked = 0, upper = 0, lower = 0;
	for (int m0=0; m0<=multistep; m0++) {
	  for (int m1=0; m1<=multistep; m1++) {
	    // Skip diagonal altogether
	    if (m0 != m1) {
	      // Only check elements with non-zero list distance
	      if (ret[m0][m1].first != ret[m0][m1].second or
		  ret[m0][m1].first != ret[m0][m1].second   ) {
		checked++;
		if (m0<m1) upper++;
		if (m0>m1) lower++;
		if (ret[m0][m1] != test[m0][m1]) differ++;
	      }
	    }
	  }
	}
	
	if (differ==0)
	  std::cout << "ZERO differences out of " << checked << " checked"
		    << " [l, u]=[" << lower << "," << upper << "]"
		    << std::endl;
	else
	  std::cout << differ << " differences out of " << checked << " checked"
		    << " [l, u]=[" << lower << "," << upper << "]"
		    << std::endl;
      }

      MPI_Barrier(MPI_COMM_WORLD);
    }
    // END: MPI rank loop
  }
  // END: deep debugging of sorted block list

  return ret;
}


void Component::CudaSortByLevel()
{
  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    // Convert from cudaParticle to a flat vector of levels.  The
    // order of the particle structures will remain fixed
    //
    cuStream->levList.resize(cuStream->cuda_particles.size());

    thrust::device_vector<cudaParticle>::iterator
      pbeg = cuStream->cuda_particles.begin(),
      pend = cuStream->cuda_particles.end();

    if (thrust_binary_search_workaround) {
      cuda_safe_call_mpi(hipStreamSynchronize(cuStream->stream),
			 __FILE__, __LINE__, myid,
			 "Component::CudaSortByLevel");
      thrust::transform(pbeg, pend, cuStream->levList.begin(), cuPartToLevel());
    } else {
      thrust::transform(exec, pbeg, pend, cuStream->levList.begin(), cuPartToLevel());
    }

    // Make room for an index
    //
    cuStream->indx1.resize(cuStream->cuda_particles.size());

    // Make the initial sequential index
    //
    thrust::sequence(cuStream->indx1.begin(), cuStream->indx1.end(), 0, 1);
  
    // First sort the keys and indices by the keys.  This gives a
    // indirect index back to the particles and a sorted levList for
    // determining the partition of the indirect index into levels
    //
    thrust::sort_by_key(cuStream->levList.begin(), cuStream->levList.end(),
			cuStream->indx1.begin());
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }

  // Direct comparison with particle vector
  // [This is expensive and only for deep debugging]
  //
  if (false) {

    bool total = true;		// Print totals in matrix rather than
				// particle end points

    for (int rank=0; rank<numprocs; rank++) {
      if (rank==myid) {
	
	std::cout << std::string(40, '-') << std::endl
		  << "---- Sorting by level debug for " << name
		  << " [" << myid << "] mstep=" << mstep
		  << " T=" << tnow << std::endl
		  << std::string(40, '-') << std::endl;
	
	// Get particles from device
	//
	thrust::host_vector<cudaParticle> pp = cuStream->cuda_particles;
	thrust::host_vector<int> lp = cuStream->levList;
	thrust::host_vector<int> li = cuStream->indx1;
	
	// Check particles from level lists
	//
	int good = 0, bad = 0;
	for (int i=0; i<li.size(); i++) {
	  int j = li[i];
	  if (pp[j].lev[0] != lp[i])
	    bad++;
	  else
	    good++;
	}
	std::cout << "---- Index check: good=" << good << " bad=" << bad << std::endl;

	// Make a map for checking indices
	//
	std::vector<int> lookup(li.size());
	for (int i=0; i<li.size(); i++) lookup[li[i]] = i;
	
	good = bad = 0;
	for (int i=0; i<li.size(); i++) {
	  int j = lookup[i];
	  if (pp[i].lev[0] != lp[j])
	    bad++;
	  else
	    good++;
	}
	
	std::cout << "---- Parts check: good=" << good << " bad=" << bad << std::endl;
	std::cout << std::string(40, '-') << std::endl;
      }

      MPI_Barrier(MPI_COMM_WORLD);
    }
    // END: MPI rank loop
  }
  // END: deep debugging of sorted block list

}



std::pair<unsigned int, unsigned int>
Component::CudaGetLevelRange(int minlev, int maxlev)
{
  std::pair<unsigned, unsigned> ret;

  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);

    // Get unsigned from input
    //
    unsigned int minl = static_cast<unsigned>(minlev);
    unsigned int maxl = static_cast<unsigned>(maxlev);

    thrust::device_vector<int>::iterator lbeg = cuStream->levList.begin();
    thrust::device_vector<int>::iterator lend = cuStream->levList.end();
    thrust::device_vector<int>::iterator lo, hi;

    if (thrust_binary_search_workaround) {
      cuda_safe_call_mpi(hipStreamSynchronize(cuStream->stream),
			 __FILE__, __LINE__, myid,
			 "Component::GetCudaLeveLRange");
      lo = thrust::lower_bound(lbeg, lend, minl);
    } else {
      lo = thrust::lower_bound(exec, lbeg, lend, minl);
    }
	
    if (thrust_binary_search_workaround) {
      cuda_safe_call_mpi(hipStreamSynchronize(cuStream->stream),
			 __FILE__, __LINE__, myid,
			 "Component::CudaGetLeveRange");
      hi = thrust::upper_bound(lbeg, lend, maxl);
    } else {
      hi = thrust::upper_bound(exec, lbeg, lend, maxl);
    }

    ret.first  = thrust::distance(lbeg, lo);
    ret.second = thrust::distance(lbeg, hi);

    if (false) {
      thrust::host_vector<int> testH(cuStream->levList);
      for (int n=0; n<10; n++) std::cout << " " << testH[n];
      std::cout << std::endl;

      std::cout << "Number of zeros="
		<< thrust::transform_reduce(cuStream->cuda_particles.begin(),
					    cuStream->cuda_particles.end(),
					    testCountLevel(0),
					    0, thrust::plus<int>())
		<< ", "
		<< thrust::transform_reduce(cuStream->levList.begin(),
					    cuStream->levList.end(),
					    testCountLevel2(0),
					    0, thrust::plus<int>())
		<< " lower=" << ret.first << " upper=" << ret.second
		<< std::endl;
    }
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 

  // Direct comparison with particle vector
  // [This is expensive and only for deep debugging]
  //
  if (false) {
    
    for (int rank=0; rank<numprocs; rank++) {
      if (rank==myid) {
	
	// Get particles from device
	//
	thrust::host_vector<cudaParticle> pp = cuStream->cuda_particles;
	thrust::host_vector<int> lp = cuStream->levList;
	thrust::host_vector<int> li = cuStream->indx1;
	
	// Make a list
	//
	std::vector<int> hl(pp.size());
	for (int p=0; p<pp.size(); p++) hl[p] = pp[p].lev[0];

	std::vector<int> indx(pp.size());
	std::iota(indx.begin(), indx.end(), 0);

	// Sort the list
	//
	struct Comparator
	{
	  Comparator(const std::vector<int> & data) : m_data(data) {}
	  bool operator()(int left, int right) const { return m_data[left] < m_data[right]; }
	  const std::vector<int> & m_data;
	};

	std::sort(indx.begin(), indx.end(), Comparator(hl));

	std::vector<int> sorted(hl.size());
	for (int p=0; p<hl.size(); p++) sorted[p] = hl[indx[p]];

	auto lot = std::lower_bound(sorted.begin(), sorted.end(), minlev);
	auto hit = std::upper_bound(sorted.begin(), sorted.end(), maxlev);

	std::cout << std::string(40, '-') << std::endl
		  << "---- Level range debug for " << name
		  << " [" << myid << "] mstep=" << mstep
		  << " T=" << tnow << std::endl;

	if (ret.first  != std::distance(sorted.begin(), lot) or
	    ret.second != std::distance(sorted.begin(), hit)   ) {
	  std::cout << std::string(40, '-') << std::endl
		    << "---- Found [" << ret.first << "," << ret.second << "]"
		    << "but expected [" << std::distance(sorted.begin(), lot)
		    << "," << std::distance(sorted.begin(), hit) << "]" << std::endl
		    << std::string(40, '-') << std::endl;
	} else {
	  std::cout << "---- Good [" << ret.first << "," << ret.second << "]"
		    << " for [" << minlev << "," << maxlev << "]"
		    << std::endl;
	}
	std::cout << std::string(40, '-') << std::endl;

      }

      MPI_Barrier(MPI_COMM_WORLD);
    }
    // END: MPI rank loop
  }
  // END: deep debugging of sorted block list

  return ret;
}

void Component::ParticlesToCuda(PartMap::iterator beg, PartMap::iterator fin)
{
  if (step_timing and use_cuda) comp->timer_cuda.start();

  auto npart = std::distance(beg, fin);
  
  // Allocate particle memory and iterators
  //
  if (host_particles.capacity()<npart) host_particles.reserve(npart);
  host_particles.resize(npart);

  cuStream->first = host_particles.begin();
  cuStream->last  = host_particles.end();

  // Translate the EXP particle to Cuda particle structures
  //
  hostPartItr hit = host_particles.begin();
  for (auto pit=beg; pit!=fin; pit++) {
    ParticleHtoD(pit->second, *(hit++));
  }

  if (step_timing and use_cuda) comp->timer_cuda.stop();
}

void Component::HostToDev(Component::cuSharedStream cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    // Resize the device array, if necessary
    //
    if (cr->cuda_particles.capacity()<npart) cr->cuda_particles.reserve(npart);
    cr->cuda_particles.resize(npart);
  
    // Copy the cuda particle structures to the device
    //
    hipMemcpyAsync(thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    thrust::raw_pointer_cast(&(*cr->first)),
		    npart*sizeof(cudaParticle),
		    hipMemcpyHostToDevice, cr->stream);

    cuda_check_last_error_mpi("hipMemcpyAsync", __FILE__, __LINE__, myid);

  }

  // Make the level index after a particle copy to device
  //
  CudaSortByLevel();
}

void Component::DevToHost(Component::cuSharedStream cr)
{
  auto npart = thrust::distance(cr->first, cr->last);
  
  if (npart) {		  // Don't bother trying to copy zero particles

    hipMemcpyAsync(thrust::raw_pointer_cast(&(*cr->first)),
		    thrust::raw_pointer_cast(&cr->cuda_particles[0]),
		    npart*sizeof(cudaParticle),
		    hipMemcpyDeviceToHost, cr->stream);

    cuda_check_last_error_mpi("hipMemcpyAsync", __FILE__, __LINE__, myid);

    hipStreamSynchronize(cr->stream);

    cuda_check_last_error_mpi("hipStreamSynchronize", __FILE__, __LINE__, myid);
  }
}


void Component::CudaToParticles(hostPartItr beg, hostPartItr end)
{
  if (step_timing and use_cuda) comp->timer_cuda.start();

  // DEBUG PRINTING (enable by setting imax>0)
  //
  const int imax = 0;
  int icnt = 0;

  // Translate the Cuda particle to the EXP particle structures
  //
  for (hostPartItr v=beg; v!=end; v++) {
    cudaParticle & p = *v;
    if (icnt < imax) {
      std::cout << "[" << icnt++ << ", " << myid << "] " << p << std::endl;
    }
    ParticleDtoH(p, particles[p.indx]);
  }

  MakeLevlist();

  if (step_timing and use_cuda) comp->timer_cuda.stop();
}

// No longer used because we need to deal with indirection
//
struct cudaZeroAcc : public thrust::unary_function<cudaParticle, cudaParticle>
{
  __host__ __device__
  cudaParticle operator()(cudaParticle& p)
  {
    for (size_t k=0; k<3; k++) p.acc[k] = 0.0;
    p.pot = p.potext = 0.0;
    return p;
  }
};

__global__ void
zeroPotAccKernel(dArray<cudaParticle> P, dArray<int> I, int stride, PII lohi)
{
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;	// Index in the stride
    int npart = i + lohi.first;	// Particle index

    if (npart < lohi.second) {

      cudaParticle & p = P._v[I._v[npart]];
      
      for (int k=0; k<3; k++) p.acc[k] = 0.0;
      p.pot = p.potext = 0.0;

    } // Particle index block
    
  } // END: stride loop
}


void Component::ZeroPotAccel(int minlev)
{
  size_t psize  = particles.size();
  
  std::pair<unsigned int, unsigned int> lohi, cur;

  if (multistep)
    lohi = CudaGetLevelRange(minlev, multistep);
  else
    lohi = {0, cuStream->cuda_particles.size()};
    
  unsigned int Ntotal = lohi.second - lohi.first;
  unsigned int Npacks = Ntotal/bunchSize + 1;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, cudaDevice);
  cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);

  // Loop over bunches
  //
  for (int n=0; n<Npacks; n++) {

    // Current bunch
    //
    cur. first = lohi.first + bunchSize*n;
    cur.second = lohi.first + bunchSize*(n+1);
    cur.second = std::min<unsigned int>(cur.second, lohi.second);
    
    if (cur.second <= cur.first) break;
    
    // Compute grid
    //
    unsigned int N         = cur.second - cur.first;
    unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
    unsigned int gridSize  = N/BLOCK_SIZE/stride;
      
    if (N > gridSize*BLOCK_SIZE*stride) gridSize++;
      

    // Pack the com values into a matrix, one particle per row
    // 
    zeroPotAccKernel<<<gridSize, BLOCK_SIZE, 0, cuStream->stream>>>
      (toKernel(cuStream->cuda_particles), toKernel(cuStream->indx1),
       stride, cur);
  }
  
}


__global__ void comKernel
(dArray<cudaParticle> P, dArray<int> I, dArray<cuFP_t> com,
 int stride, PII lohi)
{
  const int tid   = blockDim.x * blockIdx.x + threadIdx.x;

  for (int n=0; n<stride; n++) {
    int i     = tid*stride + n;
    int npart = i + lohi.first;

    if (npart < lohi.second) {

      cudaParticle & p = P._v[I._v[npart]];
      cuFP_t m = p.mass;
    
      com._v[i*10+0] = m;
      for (int k=0; k<3; k++) {
	com._v[i*10+1+k] = m * p.pos[k];
	com._v[i*10+4+k] = m * p.vel[k];
	com._v[i*10+7+k] = m * p.acc[k];
      }
    }
  }
}


// Convert linear index to row index for column reduction
//
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {

  T Ncols; // --- Number of columns
  
  __host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}
  
  __host__ __device__ T operator()(T i) { return i / Ncols; }
};

void Component::fix_positions_cuda(unsigned mlevel)
{
  (*barrier)("Container::fix_positions_cuda: BEGIN", __FILE__, __LINE__);

  const int maxBunch = 40000;

				// Zero center
  for (int i=0; i<3; i++) center[i] = 0.0;

  				// Zero variables
  mtot = 0.0;
  for (int k=0; k<dim; k++) com[k] = cov[k] = coa[k] = 0.0;

				// Zero multistep counters at and
				// above this level
  try {
    auto exec = thrust::cuda::par.on(cuStream->stream);
    
    for (int mm=mlevel; mm<=multistep; mm++) {

      hipStreamSynchronize(cuStream->stream);
      cuda_check_last_error_mpi("hipStreamSynchronize", __FILE__, __LINE__, myid);

      thrust::device_vector<int>::iterator
	lbeg = cuStream->levList.begin(), lo,
	lend = cuStream->levList.end(),   hi;

      if (thrust_binary_search_workaround) {
	hipStreamSynchronize(cuStream->stream);
	lo  = thrust::lower_bound(lbeg, lend, mm);
      } else {
	lo = thrust::lower_bound(exec, lbeg, lend, mm);
      }
      
      hipStreamSynchronize(cuStream->stream);
      cuda_check_last_error_mpi("hipStreamSynchronize", __FILE__, __LINE__, myid);

      if (thrust_binary_search_workaround) {
	hi = thrust::upper_bound(lbeg, lend, mm);
      } else {
	hi = thrust::upper_bound(exec, lbeg, lend, mm);
      }
      
      // Sort particles and get coefficient size
      //
      PII lohi = {thrust::distance(lbeg, lo), thrust::distance(lbeg, hi)};
      PII cur;
  
      unsigned int Ntotal = thrust::distance(lo, hi);
      unsigned int Npacks = Ntotal/maxBunch + 1;

      com_mas[mm] = 0.0;
      for (unsigned k=0; k<3; k++)  {
	com_lev[3*mm+k] = 0.0;
	cov_lev[3*mm+k] = 0.0;
	coa_lev[3*mm+k] = 0.0;
      }

      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, cudaDevice);
      cuda_check_last_error_mpi("hipGetDeviceProperties", __FILE__, __LINE__, myid);

      // Loop over bunches
      //
      for (int n=0; n<Npacks; n++) {

	// Current bunch
	//
	cur. first = lohi.first + maxBunch*n;
	cur.second = lohi.first + maxBunch*(n+1);
	cur.second = std::min<unsigned int>(cur.second, lohi.second);
	
	if (cur.second <= cur.first) break;
    
	// Compute grid
	//
	unsigned int N         = cur.second - cur.first;
	unsigned int stride    = N/BLOCK_SIZE/deviceProp.maxGridSize[0] + 1;
	unsigned int gridSize  = N/BLOCK_SIZE/stride;
    
	if (N > gridSize*BLOCK_SIZE*stride) gridSize++;

	// Resize storage as needed
	//
	const int Ncols = 10;	// mass, pos, vel, acc
	thrust::device_vector<cuFP_t> ret(N*Ncols);
	
	// Allocate space for row sums and indices
	//
	thrust::device_vector<cuFP_t> d_col_sums   (Ncols);
	thrust::device_vector<int>    d_col_indices(Ncols);


	// Pack the com values into a matrix, one particle per row
	// 
	comKernel<<<gridSize, BLOCK_SIZE, 0, cuStream->stream>>>
	  (toKernel(cuStream->cuda_particles), toKernel(cuStream->indx1),
	   toKernel(ret), stride, cur);

	// Perform sum over columns by summing values with equal column indices
	//
	thrust::reduce_by_key
	  (thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)),
	   thrust::make_transform_iterator(thrust::counting_iterator<int>(0), linear_index_to_row_index<int>(N)) + (N*Ncols),
	   thrust::make_permutation_iterator
	   (ret.begin(), thrust::make_transform_iterator(thrust::make_counting_iterator(0),(thrust::placeholders::_1 % N) * Ncols + thrust::placeholders::_1 / N)),
	   d_col_indices.begin(),
	   d_col_sums.begin(),
	   thrust::equal_to<int>(),
	   thrust::plus<cuFP_t>());

	// Sum the partial results
	//
	com_mas[mm] += d_col_sums[0];
	for (unsigned k=0; k<3; k++)  {
	  com_lev[3*mm+k] += d_col_sums[1+k];
	  cov_lev[3*mm+k] += d_col_sums[4+k];
	  coa_lev[3*mm+k] += d_col_sums[7+k];
	}
      }
    }
  }
  catch(thrust::system_error &e) {
    std::cerr << "Some other error happened during sort, lower_bound, or upper_bound:" << e.what() << std::endl;
    exit(-1);
  }
 
  std::vector<double> com1(3, 0.0), cov1(3, 0.0), coa1(3, 0.0);
  double              mtot1 = 0.0;

  for (unsigned mm=0; mm<=multistep; mm++) {
    for (int k=0; k<3; k++) {
      com1[k] += com_lev[3*mm + k];
      cov1[k] += cov_lev[3*mm + k];
      coa1[k] += coa_lev[3*mm + k];
    }
    mtot1 += com_mas[mm];
  }

  MPI_Allreduce(&mtot1, &mtot, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&com1[0], com, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&cov1[0], cov, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  MPI_Allreduce(&coa1[0], coa, 3, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    
  if (VERBOSE>5) {
				// Check for NaN
    bool com_nan = false, cov_nan = false, coa_nan = false;
    for (int k=0; k<3; k++)
      if (std::isnan(com[k])) com_nan = true;
    for (int k=0; k<3; k++)
      if (std::isnan(cov[k])) cov_nan = true;
    for (int k=0; k<3; k++)
      if (std::isnan(coa[k])) coa_nan = true;
    if (com_nan && myid==0)
      cerr << "Component [" << name << "] com has a NaN" << endl;
    if (cov_nan && myid==0)
      cerr << "Component [" << name << "] cov has a NaN" << endl;
    if (coa_nan && myid==0)
      cerr << "Component [" << name << "] coa has a NaN" << endl;
  }
				// Compute component center of mass and
				// center of velocity, and center of accel

  if (mtot > 0.0) {
    for (int k=0; k<dim; k++) com[k]  /= mtot;
    for (int k=0; k<dim; k++) cov[k]  /= mtot;
    for (int k=0; k<dim; k++) coa[k]  /= mtot;
  }

  if (com_system and not consp) {
    for (int k=0; k<dim; k++) com0[k] = com[k];
    for (int k=0; k<dim; k++) cov0[k] = cov[k];
  }

  if (com_system) {	   // Use local center of accel for com update
    for (int k=0; k<dim; k++) acc0[k]  = coa[k];
  } else {			// No mass, no acceleration?
    for (int k=0; k<dim; k++) acc0[k]  = 0.0;
  }

  if ((EJ & Orient::CENTER) && !EJdryrun) {
    auto ctr = orient->currentCenter();
    bool ok    = true;
    for (int i=0; i<3; i++) {
      if (std::isnan(ctr[i])) ok = false;
    } 
    if (ok) {
      for (int i=0; i<3; i++) center[i] += ctr[i];
    } else if (myid==0) {
      cout << "Orient: center failure, T=" << tnow 
	   << ", adjustment skipped" << endl;
    }
  }

  // Alternative center
  if (c0) {
    for (int i=0; i<3; i++) center[i] = c0->center[i];
  } else if (CF) {
    auto cen = (*CF)(tnow);
    for (int i=0; i<3; i++) center[i] = cen[i];
  }

  (*barrier)("Container::fix_positions_cuda: FINISH", __FILE__, __LINE__);
}


void Component::print_level_lists_cuda(double T)
{
				// Retrieve counts per level
  std::vector<int> cntr = get_level_lists_cuda();
  
  if (myid==0) {
				// Sum reduce to root
    MPI_Reduce(MPI_IN_PLACE, &cntr[0], multistep+1, MPI_INT, MPI_SUM,
	       0, MPI_COMM_WORLD);

    int tot=0;
    for (int m=0; m<=multistep; m++) tot += cntr[m];

    if (tot) {

      std::ostringstream ofil;
      ofil << outdir << runtag << ".levels";
      std::ofstream out(ofil.str().c_str(), ios::app);

      int sum=0;
      out << setw(60) << setfill('-') << '-' << endl;
      std::ostringstream sout;
      sout << "--- Component <" << name 
	   << ", " << id  << ">, T=" << T;
      out << std::setw(60) << std::left << sout.str().c_str() << std::endl;
      out << std::setw(60) << '-' << std::endl << std::setfill(' ');
      out << std::setw(3)  << "L" 
	  << std::setw(10) << "Number" 
	  << std::setw(10) << "dN/dL" 
	  << std::setw(10) << "N(<=L)"
	  << std::endl
	  << std::setw(60) << std::setfill('-') << '-'
	  << std::endl << std::setfill(' ');
      for (int n=0; n<=multistep; n++) {
	sum += cntr[n];
	out << std::setw(3)  << n 
	    << std::setw(10) << cntr[n] << std::setprecision(3) << std::fixed
	    << std::setw(10) << static_cast<double>(cntr[n])/tot
	    << std::setw(10) << static_cast<double>(sum)    /tot;
	out << std::endl;
      }
      out << std::endl << std::setw(3) << "T" << std::setw(10) << tot
	  << std::endl << std::endl << std::right;
    } else {
      std::cout << "print_level_lists_cuda [" << name 
		<< ", T=" << tnow << "]: tot=" << tot << std::endl;
    }

  } else {
				// Sum reduce counts to root
    MPI_Reduce(&cntr[0], 0, multistep+1, MPI_INT, MPI_SUM,
	       0, MPI_COMM_WORLD);
  }

}

std::vector<int> Component::get_level_lists_cuda()
{
				// Retrieve counts per level
  std::vector<int> cntr(multistep+1);
  for (int m=0; m<=multistep; m++) {
    cntr[m] = thrust::transform_reduce(cuStream->cuda_particles.begin(),
				       cuStream->cuda_particles.end(),
				       testCountLevel(m),
				       0, thrust::plus<int>());
  }

  return cntr;
}


// No cuda code here but only used after CudaToParticles() call for
// testing
void Component::MakeLevlist()
{
  levlist.resize(multistep+1);
  for (auto & v : levlist) v.clear();
  for (auto & v : particles) levlist[v.second->level].push_back(v.first);
}
